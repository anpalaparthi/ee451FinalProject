// https://towardsdatascience.com/implementing-svm-from-scratch-784e4ad0bc6a
// Implementation based off of python SVM techniques from this medium article

// Basic setup/initialization of weights and biases (w, b)
// Map the class labels from {0,1} to {-1,1}
// Perform gradient descent for n iterations, which involves the computation
//      of gradients and updated the weights an biases accordingly
// Make the final prediction


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>

using namespace std;

#define NUM_PIXELS 7500

//NUM_FEATURES = lenW = 2
// kernel<<<dimGrid, (numIters, size, lenW, gpuW, b, gpuCLS, gpuX, lr, lambdaParam);				

__global__ void kernel(int numIters, int size, int lenW, double* w, double* b, double* cls, double* X, double lr, double lambdaParam, double* x) {

    int id = threadIdx.x;
    double wVal = 0;
    // __shared__ double myW[NUM_PIXELS];
    __shared__ bool constraint;
    __shared__ double myB;
    // __shared__ double x[NUM_PIXELS];

    // myW[id] = 0;
    myB = 0;
    // if (id == 0) {
    // printf("id = %d, digit = %d, myB = %f\n", id, digit, myB);
    // }

    // for (int i = 0; i < 8400; i++) {
    //     if (i % 1000 == 0) {
    //         printf("id = %d, i = %d, X[i] = %f\n", id, i, X[i]);
    //     }
    // }

    double result = 0;
    double xVal = 0;
    for (int i = 0; i < numIters; i++) {
        for (int j = 0; j < size; j++) {
            //check constraint: dotProduct with recursive doubling
            
            xVal = X[(j * NUM_PIXELS) + id];

            // xVal = X[(j * BAD_PIXELS) + id];
            // x[id] = xVal * myW[id];
            x[id] = xVal * wVal;
            __syncthreads();
            /*
            for (int k = 2; k <= MORE_PIXELS; k *= 2) {
            // for (int k = 2; k <= BAD_PIXELS; k *= 2) {
                __syncthreads();
                if ((id % k) == 0) {
                    // printf("DOUBLING k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);

                    if ((id + (k/2)) < NUM_PIXELS) {
                        x[id] += x[id + (k/2)];
                        // printf("DOUBLING k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);
                    } 
                    // else {
                    //     printf("NOOOOO k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);
                    // }
                }
            }
            */

            if (id == 0) {
                // result = x[id] + myB;
                result = 0;
                for (int k = 0; k < NUM_PIXELS; k++) {
                    result += x[k];
                }
                result += myB;

                if ((cls[j] * result) >= 1) {
                    constraint = true;
                } else {
                    constraint = false;
                    myB -= lr * (-cls[j]);
                }
                
            }
            __syncthreads();

            //get and update gradients
            if (constraint) {
                // myW[id] = myW[id] - (lr * myW[id] * lambdaParam);
                wVal = wVal - (lr * wVal * lambdaParam);
            } else {
                // myW[id] = myW[id] - (lr * ((lambdaParam * myW[id]) - (cls[j] * xVal)));
                wVal = wVal - (lr * ((lambdaParam * wVal) - (cls[j] * xVal)));
            }
            
        }
        __syncthreads();
    }
    
    w[id] = wVal;
    if (id == 0) {
        b[0] = myB;
    }
    // printf("digit = %d, id = %d, end w[id] = %f, b = %f\n", digit, id, w[id], b[0]);
    
}

class SVM {

    private:
        double lr;
        double lambdaParam;
        int numIters;
        double* w;
        int lenW;
        double b;
        double* clsMap;

            
        void initWeightsBias(int numFeatures) { //self, X
            w = new double[numFeatures];
            for (int i = 0; i < numFeatures; i++) {
                w[i] = 0;
            }
            lenW = numFeatures;
            b = 0;
        }

        void getClsMap(int* y, int size) { //self, y
            if (clsMap == NULL) {
                clsMap = new double[size];
            }
            for (int i = 0; i < size; i++) {
                if (y[i] == 0) {
                    clsMap[i] = -1;
                } else {
                    clsMap[i] = 1;
                }
            }
        }

        // a and b must be the same length
        double dotProduct(double* a, double* b, int size) {
            double sum = 0;
            for (int i = 0; i < size; i++) {
                // if (i % 100 == 0) {
                //     cout << "dot product i = " << i << ", a[i] = " << a[i] << ", b[i] = " << b[i] << endl;
                // }
                sum += a[i] * b[i];
            }
            return sum;
        }

        // GPU
        bool satisfyConstraint(double* x, int idx) { //self, x, idx
            // dot product linearModel = x * w + b
            double linearModel = dotProduct(x, w, lenW);
            linearModel += b;

            if ( (clsMap[idx] * linearModel) >= 1) {
                return true;
            }
            return false;
        }

        // FREE dw
        void getGradients(bool constraint, double* x, int idx, double*& dw, double& db) { //self, constrain, x, idx
            dw = new double[lenW];
            if (constraint) {
                for (int i = 0; i < lenW; i++) {
                    dw[i] = (w[i] * lambdaParam);
                }
                db = 0;
                return;
            }

            for (int i = 0; i < lenW; i++) {
                dw[i] = (lambdaParam * w[i]) - (clsMap[idx] * x[i]);
            }
            db = -clsMap[idx];
        }

        void updateWeights(double* dw, double db) { //self, dw, db
            for (int i = 0; i < lenW; i++) {
                w[i] -= lr * dw[i];
            }
            b -= lr * db;
        } 

    public:
        SVM(double learningRate, double lamba, double iters) {
            lr = learningRate;
            lambdaParam = lamba;
            numIters = iters;
            w = NULL;
            b = 0;
            clsMap = NULL;
        }

        ~SVM() {
            if (w != NULL) {
                printf("free memory w\n");
                delete[] w;
            }
            if (clsMap != NULL) {
                printf("free memory clsMap\n");
                delete[] clsMap;
            }
        }

        //size = length of y, length of X (X dims = [size][numFeatures])
        void fit(double* X, int numFeatures, int* y, int size) { //self, X, y
            
            initWeightsBias(numFeatures);
            getClsMap(y, size); // updates clsMap

            double* gpuX;
            double* gpuCLS;
            double* gpuW;
            double* gpuB;
            double* gpuXGlobal;

            double* bArr = new double[1];
            bArr[0] = b;

	        hipMalloc((void**)&gpuX, sizeof(double)*size*numFeatures); 
	        hipMalloc((void**)&gpuCLS, sizeof(double)*size); 
	        hipMalloc((void**)&gpuW, sizeof(double)*numFeatures);  
	        hipMalloc((void**)&gpuB, sizeof(double));   
	        hipMalloc((void**)&gpuXGlobal, sizeof(double)*NUM_PIXELS); 
            struct timespec start, stop; 
            double time;
            if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
            
	        hipMemcpy(gpuX, X, sizeof(double)*size*numFeatures, hipMemcpyHostToDevice);
	        hipMemcpy(gpuCLS, clsMap, sizeof(double)*size, hipMemcpyHostToDevice);
	        hipMemcpy(gpuW, w, sizeof(double)*numFeatures, hipMemcpyHostToDevice);
            hipMemcpy(gpuB, bArr, sizeof(double), hipMemcpyHostToDevice);

            //kernel(int numIters, int size, int lenW, double* w, double b, double* cls, double* X) {
            // cout << "lenW = " << lenW << endl;
            dim3 dimGrid(1);
            dim3 dimBlock(lenW);
            //__global__ void kernel(int numIters, int size, int lenW, double* w, double* b, double* cls, double* X, double lr, double lambdaParam, int digit) {

            kernel<<<dimGrid, dimBlock>>>(numIters, size, numFeatures, gpuW, gpuB, gpuCLS, gpuX, lr, lambdaParam, gpuXGlobal);				
 
            
            // cudaMemcpy(X, gpuX, sizeof(double)*size*numFeatures, cudaMemcpyDeviceToHost);
	        // cudaMemcpy(clsMap, gpuCLS, sizeof(double)*size, cudaMemcpyDeviceToHost);
	        hipMemcpy(w, gpuW, sizeof(double)*numFeatures, hipMemcpyDeviceToHost);
	        hipMemcpy(bArr, gpuB, sizeof(double), hipMemcpyDeviceToHost);

            if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
            time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
            printf("time is %f ns\n", time*1e9);	
            
            b = bArr[0];
            // printf("after kernel bArr = %f, b = %f\n", bArr[0], b);
            delete[] bArr;

            hipFree(gpuX);  
            hipFree(gpuCLS);  
            hipFree(gpuW);  
            hipFree(gpuB);
            hipFree(gpuXGlobal);

        }

        //X is of dims [size][numFeatures]
        // FREE estimate
        int* predict(double** X, int size) { //self, X
            // printf("predict w[id] = %f, b = %f\n", w[0], b);
            int* estimate = new int[size]; //hold dot products
            for (int i = 0; i < size; i++) {
                estimate[i] = dotProduct(X[i], w, lenW);
                estimate[i] += b;
            }

            //predict class labels
            for (int i = 0; i < size; i++) {
                if (estimate[i] < 0) {
                    estimate[i] = 0;
                } else {
                    estimate[i] = 1;
                }
            }

            cout << "weight: " << w[0] << ", " << w[34] << ", " << w[1450] << ", " << w[2354] << ", " << w[6458] << endl; 
            return estimate;
        }

};

// both yTrue and yPred have size elements
double accuracy(int* yTrue, int* yPred, int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        if (i % 100 == 0) {
            cout << "yTrue " << yTrue[i] << " | " << "yPred " << yPred[i] << endl;
        }
        if (yTrue[i] == yPred[i]) {
            sum++;
        }
    }
    return (sum / (1.0 * size));
}

void parseMNISTData(string dataFileStr, int numTrain, int numTest, double** Xtrain, int* ytrain, double** Xtest, int* ytest) {
    ifstream inputFile;
    inputFile.open(dataFileStr);
    cout << "open file" << endl;
    
    if (!inputFile) {
        cout << "AAAAAAA file not open" << endl;
    } else {
        cout << "BBBBBBB file opened" << endl;
    }

    string line = "";
    int total = 0;
    bool flag = true;
    int idx = 0;
    while (getline(inputFile, line)) {
        // if (flag) {
        //     flag = false;
        //     continue;
        // }
        int label;
        double pixels[NUM_PIXELS];
        string temp = "";

        stringstream inputString(line);
        // ss >> xData1 >> xData2 >> cls;
        getline(inputString, temp, ',');
        label = atoi(temp.c_str());
        for (int i = 0; i < NUM_PIXELS; i++) {
            getline(inputString, temp, ',');
            pixels[i] = atof(temp.c_str());
        }        

        if (total == numTrain) {
            idx = 0;
        }
        // cout << "total = " << total << " | numTrain = " << numTrain << " | numTest = " << numTest << " | idx = " << idx << endl;
        // cout << "xData1 = " << xData1 << " | xData2 = " << xData2 << " | cls = " << cls << endl;
        if (total < numTrain) {
            for (int i = 0; i < NUM_PIXELS; i++) {
                Xtrain[idx][i] = pixels[i];
            }
            ytrain[idx] = label;
        } else {
            for (int i = 0; i < NUM_PIXELS; i++) {
                Xtest[idx][i] = pixels[i];
            }
            ytest[idx] = label;
        }

        line = "";
        total++;
        idx++;

        if (total == (numTrain + numTest)) {
            break;
        }

    }
        
    cout << "file read" << endl;
    inputFile.close();
    cout << "file closed" << endl;

}


void parseBlobData(string dataFileStr, double** Xtrain, int* ytrain, int numTrain, double** Xtest, int* ytest) {
    ifstream inputFile;
    inputFile.open(dataFileStr);
    cout << "open file" << endl;
    
    string line = "";
    int total = 0;
    bool flag = true;
    int idx = 0;
    while (getline(inputFile, line)) {
        if (flag) {
            flag = false;
            continue;
        }
        double xData1;
        double xData2;
        int cls;
        string temp = "";

        stringstream inputString(line);
        // ss >> xData1 >> xData2 >> cls;
        getline(inputString, temp, ',');
        xData1 = atof(temp.c_str());
        getline(inputString, temp, ',');
        xData2 = atof(temp.c_str());
        getline(inputString, temp, ',');
        cls = atoi(temp.c_str());

        if (total == numTrain) {
            idx = 0;
        }
        cout << "total = " << total << " | numTrain = " << numTrain << " | idx = " << idx << endl;
        cout << "xData1 = " << xData1 << " | xData2 = " << xData2 << " | cls = " << cls << endl;
        if (total < numTrain) {
            Xtrain[idx][0] = xData1;
            Xtrain[idx][1] = xData2;
            ytrain[idx] = cls;
        } else {
            Xtest[idx][0] = xData1;
            Xtest[idx][1] = xData2;
            ytest[idx] = cls;
        }

        line = "";
        total++;
        idx++;

    }
        
    cout << "file read" << endl;
    inputFile.close();
    cout << "file closed" << endl;

}


int main() {

    cout << "start" << endl;
    // training/test data parameters
    int numSamples = 40000;
    double testSize = 0.3;
    double trainSize = 0.2;
    int numTrain = trainSize * numSamples;
    int numTest = testSize * numSamples;
    int numFeatures = NUM_PIXELS;

    // SVM hyperparameters
    double learningRate = 0.001; //1e-3
    double lamba = 0.01; //1e-2 
    // double iters = 1000; 
    double iters = 100;
    
    cout << "defined params" << endl;

    //allocate memory for training and test data
    double** Xtrain = new double*[numTrain];
    int* ytrain = new int[numTrain];
    for (int i = 0; i < numTrain; i++) {
        Xtrain[i] = new double[numFeatures];
    }
    
    double** Xtest = new double*[numTest];
    int* ytest = new int[numTest];
    for (int i = 0; i < numTest; i++) {
        Xtest[i] = new double[numFeatures];
    }

    
    cout << "finished allocation" << endl;
    cout << "numTrain = " << numTrain << " | numTest = " << numTest << endl;

        // read from csv: https://www.youtube.com/watch?v=NFvxA-57LLA
    string dataFileStr = "idc_dataset40k_shuffled.csv";

    if (dataFileStr == "idc_dataset40k_shuffled.csv") {
        parseMNISTData(dataFileStr, numTrain, numTest, Xtrain, ytrain, Xtest, ytest);
    } else {
        cout << "File " << dataFileStr << " not supported" << endl;
    }

    double* Xtrain1D = new double[numTrain * numFeatures];
    for (int i = 0; i < numTrain; i++) {
        for (int j = 0; j < numFeatures; j++) {
            Xtrain1D[(i * numFeatures) + j] = Xtrain[i][j];
        }
    }

    SVM classifier = SVM(learningRate, lamba, iters);
    classifier.fit(Xtrain1D, numFeatures, ytrain, numTrain);
    int* predictions = classifier.predict(Xtest, numTest);
    
    
    cout << "cassifier trained " << endl;
    double acc = accuracy(ytest, predictions, numTest);
    printf("SVM Accuracy: %f\n", acc);

    delete[] predictions;
    delete[] Xtrain1D;

    //free memory of training and test data
    for (int i = 0; i < numTrain; i++) {
        delete[] Xtrain[i];
    }
    delete[] Xtrain;
    delete[] ytrain;
    
    for (int i = 0; i < numTest; i++) {
        delete[] Xtest[i];
    }
    delete[] Xtest;
    delete[] ytest;

    cout << "done" << endl;
    
}