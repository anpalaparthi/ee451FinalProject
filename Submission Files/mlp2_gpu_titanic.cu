// Based on python tutorial
// https://www.kaggle.com/code/vitorgamalemos/multilayer-perceptron-from-scratch



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>
#include <cmath>
#include <random>

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>

#define SEED 100

#define NUM_CLASSES 2
#define NUM_FEATURES 8
#define NUM_PIXELS 8 // notused
using namespace std;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

// number of threads = hiddenLayer
// each thread computes 1 element for outputL1[]
// x = (1 x inputLayer), weightHidden = (inputLayer x hiddenLayer)
// outputL1 = 1 x hiddenLayer
// outputL1[id] = dot product x and weightHidden[:][id] (: from 0 to inputLayer)
// outputL1[id] = sigmoid(outputL1[id] + biasHidden[id])
//   sigmoid = 1.0 / (1.0 + exp(-1.0 * val))
__global__ void kernelOutputL1(double* x, double* weightHidden, double* biasHidden, double* outputL1, int inputLayer, int hiddenLayer) {
    int id = threadIdx.x;
    double val = 0;
    for (int i = 0; i < inputLayer; i++) {
        val = val + ((1.0 * x[i]) * weightHidden[(i * hiddenLayer) + id]);
    }
    val = val + biasHidden[id];
    outputL1[id] = 1.0 / (1.0 + exp(-1.0 * val));
    __syncthreads();
}

// number of threads = outputLayer
// each thread computes 1 element of outputL2[]
// outputL1 = (1 x hiddenLayer), weightOutput = (hiddenlayer x outputLayer)
// outputL2 = 1 x outputLayer
// outputL2[id] = dot product outputL1 and weightOutput[:][id] (: j from 0 to hiddenLayer)
// outputL2[id] = sigmoid (outputL2[id] + biasOutput[id])
//   sigmoid = 1.0 / (1.0 + exp(-1.0 * val))
__global__ void kernelOutputL2(double* outputL1, double* outputL2, double* weightOutput, double* biasOutput, int hiddenLayer, int outputLayer, int yVal, int* output) {
    int id = threadIdx.x;
    double val = 0;
    for (int i = 0; i < hiddenLayer; i++) {
        val = val + (outputL1[i] * weightOutput[(i * outputLayer) + id]);
    }
    val = val + biasOutput[id];
    outputL2[id] = 1.0 / (1.0 + exp(-1.0 * val));

    if (id == yVal) {
        output[id] = 1;
    } else {
        output[id] = 0;
    }
    printf("id = %d, output[%d] = %d\n", id, id, output[id]);
    __syncthreads();
}

// number of threads = outputLayer
// each thread computes deltaOutput[id], weightOutput[i][id], biasOutput[id]
__global__ void kernelUpdateWeightOutput(int* output, double* outputL1, double* outputL2, double* weightOutput, double* biasOutput, double* deltaOutput, double lr, int hiddenLayer, int outputLayer) {
    int id = threadIdx.x;
    double outputL2Val = outputL2[id];
    double deltaVal = (1.0 * output[id]) - outputL2Val;
    deltaVal = (-1.0) * deltaVal - (outputL2Val * (1.0 - outputL2Val));

    for (int i = 0; i < hiddenLayer; i++) {
        weightOutput[(i * outputLayer) + id] = weightOutput[(i * outputLayer) + id] - (lr * deltaVal * outputL1[i]);
    }
    // biasOutput[id] -= (lr * deltaVal * hiddenLayer * outputLayer);
    biasOutput[id] = biasOutput[id] - (lr * deltaVal);
    deltaOutput[id] = deltaVal;
    __syncthreads();
}

// number of threads = hiddenLayer
// each thread computes product[id] (implicit), deltaHidden[id] (implicit), weightHidden[i][id], biasHidden[id]
__global__ void kernelUpdateWeightHidden(double* input, double* outputL1, double* weightOutput, double* weightHidden, double* biasHidden, double* deltaOutput, double lr, int inputLayer, int hiddenLayer, int outputLayer) {
    int id = threadIdx.x;
    double outputL1Val = outputL1[id];
    double deltaVal;
    /*
    double* product = new double[hiddenLayer * 1];
    for (int i = 0; i < hiddenLayer; i++) {
        product[i] = 0;
        for (int k = 0; k < outputLayer; k++) {
            product[i] += weightOutput[(i * outputLayer) + k] + deltaOutput[k];
        }
    }
    */
    double productVal = 0;
    for (int k = 0; k < outputLayer; k++) {
        productVal = productVal + (weightOutput[(id * outputLayer) + k] + deltaOutput[k]);
    }

    deltaVal = productVal * (outputL1Val * (1.0 - outputL1Val));

    for (int i = 0; i < inputLayer; i++) {
        weightHidden[(i * hiddenLayer) + id] = weightHidden[(i * hiddenLayer) + id] - (lr * deltaVal * input[i]);
    }
    // biasHidden[id] -= (lr * deltaVal * inputLayer * hiddenLayer);
    biasHidden[id] = biasHidden[id] - (lr * deltaVal);
    __syncthreads();
}

class MLP {
    private:
        int inputLayer;
        int hiddenLayer;
        int outputLayer;
        double lr;
        int maxEpochs;
        double biasHiddenValue;
        double biasOutputValue;

        double* weightHidden; // inputLayer x hiddenLayer
        double* weightOutput; // hiddenLayer x outputLayer
        double* biasHidden; //hiddenLayer x 1
        double* biasOutput; // outputLayer x 1
        int numClasses;

    public:
        MLP (int inputLayer_in,
            int hiddenLayer_in,
            int outputLayer_in,
            double learningRate_in,
            int maxEpochs_in,
            double biasHiddenValue_in,
            double biasOutputValue_in,
            int numClasses_in) 
            {
                inputLayer = inputLayer_in;
                hiddenLayer = hiddenLayer_in;
                outputLayer = outputLayer_in;
                lr = learningRate_in;
                maxEpochs = maxEpochs_in;
                biasHiddenValue = biasHiddenValue_in;
                biasOutputValue = biasOutputValue_in;
                numClasses = numClasses_in;

                //initialize weights
                weightHidden = new double[inputLayer * hiddenLayer]; // inputLayer x hiddenLayer
                weightOutput = new double[hiddenLayer * outputLayer]; // hiddenLayer x outputLayer
                biasHidden = new double[hiddenLayer]; //hiddenLayer x 1
                biasOutput = new double[outputLayer]; // outputLayer x 1

                srand(SEED);
                //init weightHidden
                for (int i = 0; i < inputLayer; i++) {
                    for (int j = 0; j < hiddenLayer; j++) {
                        // weightHidden[(i * hiddenLayer) + j] = (2.0 * ((double) rand() / (RAND_MAX))) - 1;
                        weightHidden[(i * hiddenLayer) + j] = ((double) rand() / (RAND_MAX)) - 1;
                    }
                }

                // init weightOutput
                for (int i = 0; i < hiddenLayer; i++) {
                    for (int j = 0; j < outputLayer; j++) {
                        // weightOutput[(i * outputLayer) + j] = (2.0 * ((double) rand() / (RAND_MAX))) - 1;
                        weightOutput[(i * outputLayer) + j] = ((double) rand() / (RAND_MAX)) - 1;
                    }
                }

                //init biasHidden
                for (int i = 0; i < hiddenLayer; i++) {
                    biasHidden[i] = biasHiddenValue;
                }
                
                //init biasOutput
                for (int i = 0; i < outputLayer; i++) {
                    biasOutput[i] = biasOutputValue;
                }

                // //print weightHidden
                // for (int i = 0; i < inputLayer; i++) {
                //     for (int j = 0; j < hiddenLayer; j++) {
                //         cout << weightHidden[(i * hiddenLayer) + j] << endl;
                //     }
                // }

                // // init weightOutput
                // for (int i = 0; i < hiddenLayer; i++) {
                //     for (int j = 0; j < outputLayer; j++) {
                //         cout << weightOutput[(i * outputLayer) + j] << endl;
                //     }
                // }
            }

        ~MLP() {
            delete[] weightHidden; // inputLayer x hiddenLayer
            delete[] weightOutput; // hiddenLayer x outputLayer
            delete[] biasHidden; //hiddenLayer x 1
            delete[] biasOutput; // outputLayer x 1
        }

        //val needs to be outputted by a sigmoid
        double derivativeSingle(double val) {
            return (val * (1.0 - val));
        }

        double sigmoidSingle(double val) {
            return 1.0 / (1.0 + exp(-1.0 * val));
        }
        
        // input = inputLayer x 1
        // output = outputLayer x 1
        // outputL1 = hiddenLayer x 1
        // outputL2 = outputLayer x 1
        void backPropagation(double* input, double* output, double* outputL1, double* outputL2) {
            // Error output layer
            double* deltaOutput = new double[outputLayer];
            for (int i = 0; i < outputLayer; i++) {
                //errorOutput = output - outputL2
                deltaOutput[i] = output[i] - outputL2[i];

                //deltaOutput = -1*errorOutput*deriv(outputL2)
                deltaOutput[i] = (-1.0) * deltaOutput[i] - derivativeSingle(outputL2[i]);
            }

            //update weights outputLayer and hiddenLayer
            for (int i = 0; i < hiddenLayer; i++) {
                for (int j = 0; j < outputLayer; j++) {
                    weightOutput[(i * outputLayer) + j] -= (lr * deltaOutput[j] * outputL1[i]);
                    biasOutput[j] -= (lr * deltaOutput[j]);
                }
            }

            // hidden layer
            // matmul weightOutput x deltaOutput 
            // = (hiddenLayer x outputLayer) x (outputLayer x 1)
            // = hiddenLayer x 1
            double* product = new double[hiddenLayer * 1];
            for (int i = 0; i < hiddenLayer; i++) {
                for (int j = 0; j < 1; j++) {
                    product[(i * 1) + j] = 0;
                    for (int k = 0; k < outputLayer; k++) {
                        product[(i * 1) + j] += weightOutput[(i * outputLayer) + k] + deltaOutput[(k * 1) + j];
                    }
                }
            }

            // product x deriv(outputL1) 
            // = (hiddenLayer x 1) x (hiddenLayer x 1) (element wise)
            double* deltaHidden = new double[hiddenLayer];
            for (int i = 0; i < hiddenLayer; i++) {
                deltaHidden[i] = product[i] * derivativeSingle(outputL1[i]);
            }

            //update weights hidden layer and input layer
            for (int i = 0; i < inputLayer; i++) {
                for (int j = 0; j < hiddenLayer; j++) {
                    weightHidden[(i * hiddenLayer) + j] -= (lr * deltaHidden[j] * input[i]);
                    biasHidden[j] -= (lr * deltaHidden[j]);
                }
            }

            delete[] deltaOutput;
            delete[] product;
            delete[] deltaHidden;            
        }
        
        void fit(double* xtrain, int* ytrain, int numSamples) {
            // double* output = new double[numClasses]; // numClasses = outputLayer
            // double* outputL1 = new double[hiddenLayer];
            // double* outputL2 = new double[outputLayer];

            int* gpuOutput;
            double* gpuOutputL1;
            double* gpuOutputL2;
            double* gpuXtrain;
            double* gpuWeightHidden;
            double* gpuWeightOutput;
            double* gpuBiasHidden;
            double* gpuBiasOutput;
            double* gpuDeltaOutput;

            hipMalloc((void**)&gpuOutput, sizeof(int)*numClasses);
            hipMalloc((void**)&gpuOutputL1, sizeof(double)*hiddenLayer);
            hipMalloc((void**)&gpuOutputL2, sizeof(double)*outputLayer);
            hipMalloc((void**)&gpuXtrain, sizeof(double)*numSamples*inputLayer);
            hipMalloc((void**)&gpuWeightHidden, sizeof(double)*inputLayer*hiddenLayer); 
            hipMalloc((void**)&gpuWeightOutput, sizeof(double)*hiddenLayer*outputLayer); 
            hipMalloc((void**)&gpuBiasHidden, sizeof(double)*hiddenLayer); 
            hipMalloc((void**)&gpuBiasOutput, sizeof(double)*outputLayer); 
            hipMalloc((void**)&gpuDeltaOutput, sizeof(double)*outputLayer); 

            struct timespec start, stop; 
            double time;
            if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
            

            hipMemcpy(gpuXtrain, xtrain, sizeof(double)*numSamples*inputLayer, hipMemcpyHostToDevice);
            hipMemcpy(gpuWeightHidden, weightHidden, sizeof(double)*inputLayer*hiddenLayer, hipMemcpyHostToDevice);
            hipMemcpy(gpuWeightOutput, weightOutput, sizeof(double)*hiddenLayer*outputLayer, hipMemcpyHostToDevice);
            hipMemcpy(gpuBiasHidden, biasHidden, sizeof(double)*hiddenLayer, hipMemcpyHostToDevice);
            hipMemcpy(gpuBiasOutput, biasOutput, sizeof(double)*outputLayer, hipMemcpyHostToDevice);

            dim3 dimGrid(1);
            dim3 dimBlockHidden(hiddenLayer);
            dim3 dimBlockOutput(outputLayer);

            for (int epoch = 0; epoch < maxEpochs; epoch++) {
                if (epoch % 50 == 0) {
                    cout << "epoch = " << epoch << endl;
                }
                for (int sample = 0; sample < numSamples; sample++) {
                    //Forward propagation
                    // x = &xtrain[sample * inputLayer];
                    kernelOutputL1<<<dimGrid, dimBlockHidden>>>(&gpuXtrain[sample * inputLayer], gpuWeightHidden, gpuBiasHidden, gpuOutputL1, inputLayer, hiddenLayer);
                    kernelOutputL2<<<dimGrid, dimBlockOutput>>>(gpuOutputL1, gpuOutputL2, gpuWeightOutput, gpuBiasOutput, hiddenLayer, outputLayer, ytrain[sample], gpuOutput);

                    //backprop
                    kernelUpdateWeightOutput<<<dimGrid, dimBlockOutput>>>(gpuOutput, gpuOutputL1, gpuOutputL2, gpuWeightOutput, gpuBiasOutput, gpuDeltaOutput, lr, hiddenLayer, outputLayer);
                    kernelUpdateWeightHidden<<<dimGrid, dimBlockHidden>>>(&gpuXtrain[sample * inputLayer], gpuOutputL1, gpuWeightOutput, gpuWeightHidden, gpuBiasHidden, gpuDeltaOutput, lr, inputLayer, hiddenLayer, outputLayer);
                }
            }

            hipMemcpy(weightHidden, gpuWeightHidden, sizeof(double)*inputLayer*hiddenLayer, hipMemcpyDeviceToHost);
            hipMemcpy(weightOutput, gpuWeightOutput, sizeof(double)*hiddenLayer*outputLayer, hipMemcpyDeviceToHost);
            hipMemcpy(biasHidden, gpuBiasHidden, sizeof(double)*hiddenLayer, hipMemcpyDeviceToHost);
            hipMemcpy(biasOutput, gpuBiasOutput, sizeof(double)*outputLayer, hipMemcpyDeviceToHost);
            
            if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
            time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
            printf("time is %f sec\n", time);	

            hipFree(gpuOutput);
            hipFree(gpuOutputL1);
            hipFree(gpuOutputL2);
            hipFree(gpuXtrain);
            hipFree(gpuWeightHidden); 
            hipFree(gpuWeightOutput); 
            hipFree(gpuBiasHidden); 
            hipFree(gpuBiasOutput); 
            hipFree(gpuDeltaOutput); 
        }

        //prediction = empty array allocated for size = num * numClasses
        /*
        void predict(double* xtest, int* prediction, int num) {
            double* outputL1 = new double[hiddenLayer];
            double* outputL2 = new double[outputLayer];
            double* x;
            int* p;
            for (int sample = 0; sample < num; sample++) {
                //Forward propagation
                x = &xtest[sample * inputLayer];
                p = &prediction[sample * numClasses];
                // find outputL1 = sigmoid(input x weightHidden + biasHidden.T)
                // (input is transposed to 1 x inputLayer, weightHidden = inputLayer x hiddenLayer)
                // input x weightHidden (1 x inputLayer) x (inputLayer x hiddenLayer)
                for (int i = 0; i < hiddenLayer; i++) {
                    outputL1[i] = 0;
                    for (int j = 0; j < inputLayer; j++) {
                        outputL1[i] += x[j] * weightHidden[(j * hiddenLayer) + i];
                    }
                    outputL1[i] = sigmoidSingle(outputL1[i] + biasHidden[i]);
                }

                // find outputL2 = sigmoid(outputL1 x weightOutput + biasOutput.T)
                // outputL1 = (transposed) 1 x hiddenLayer
                // weightOutput = hiddenLayer x outputLayer
                // outputL2 = (1 x hiddenLayer) x (hiddenLayer x outputLayer) = 1 x outputLayer
                for (int i = 0; i < outputLayer; i++) {
                    outputL2[i] = 0;
                    for (int j = 0; j < hiddenLayer; j++) {
                        outputL2[i] += outputL1[j] * weightOutput[(j * outputLayer) + i];
                    }
                    outputL2[i] = sigmoidSingle(outputL2[i] + biasOutput[i]);
                }

                double max = -1;
                int maxId = -1;
                for (int i = 0; i < numClasses; i++) {
                    if (outputL2[i] > max) {
                        max = outputL2[i];
                        maxId = i;
                    }
                }
                
                // one-hot encoding
                // for (int i = 0; i < numClasses; i++) {
                //     output[i] = 0;
                // }
                // output[(int)(y[sample])] = 1;
                if (maxId == 0) {
                    p[0] = 1;
                    p[1] = 0;
                } else {
                    p[0] = 0;
                    p[1] = 1;
                }
                cout << "maxId = " << maxId << ", outputL2[0] = " << outputL2[0] << ", outputL2[1] = " << outputL2[1] << endl;  
            }

            delete[] outputL1;
            delete[] outputL2;
        }
        */
       void predict(double* xtest, int* prediction, int num) {
            //forward propogation
            double* outputL1 = new double[hiddenLayer];
            double* outputL2 = new double[outputLayer];
            double* x;
            int* p;
            for (int sample = 0; sample < num; sample++) {
                //Forward propagation
                x = &xtest[sample * inputLayer];
                p = &prediction[sample * numClasses];
                // find outputL1 = sigmoid(input x weightHidden + biasHidden.T)
                // (input is transposed to 1 x inputLayer, weightHidden = inputLayer x hiddenLayer)
                // input x weightHidden (1 x inputLayer) x (inputLayer x hiddenLayer)
                if (sample % 2 == 0) {
                    cout << "outputL1 pre sigmoid: " ;
                }
                for (int i = 0; i < hiddenLayer; i++) {
                    outputL1[i] = 0;
                    for (int j = 0; j < inputLayer; j++) {
                        outputL1[i] += x[j] * weightHidden[(j * hiddenLayer) + i];
                    }
                    if (sample % 2 == 0) {
                        cout << outputL1[i] << " ";
                    }
                    outputL1[i] = sigmoidSingle(outputL1[i] + biasHidden[i]);
                }
                if (sample % 2 == 0) {
                    cout << endl;
                }

                // find outputL2 = sigmoid(outputL1 x weightOutput + biasOutput.T)
                // outputL1 = (transposed) 1 x hiddenLayer
                // weightOutput = hiddenLayer x outputLayer
                // outputL2 = (1 x hiddenLayer) x (hiddenLayer x outputLayer) = 1 x outputLayer
                for (int i = 0; i < outputLayer; i++) {
                    outputL2[i] = 0;
                    for (int j = 0; j < hiddenLayer; j++) {
                        outputL2[i] += outputL1[j] * weightOutput[(j * outputLayer) + i];
                    }
                    outputL2[i] = sigmoidSingle(outputL2[i] + biasOutput[i]);
                }

                double max = -1;
                int maxId = -1;
                for (int i = 0; i < outputLayer; i++) {
                    if (outputL2[i] > max) {
                        max = outputL2[i];
                        maxId = i;
                    }
                }
                
                // one-hot encoding
                // for (int i = 0; i < outputLayer; i++) {
                //     p[i] = 0;
                // }
                // p[maxId] = 1;
                if (maxId == 0) {
                    p[0] = 1;
                    p[1] = 0;
                } else {
                    p[0] = 0;
                    p[1] = 1;
                }
                
                
                if (sample % 2 == 0) {
                    cout << "x: ";
                    for (int i = 0; i < inputLayer; i++) {
                        cout << x[i] << " " ;
                    }
                    cout << endl;
                    cout << "maxId = " << maxId;
                    for (int i = 0; i < outputLayer; i++) {
                        cout << ", outputL2[" << i << "] = " << outputL2[i];
                    }
                    cout << endl;  
                    cout << "outputL1: " ;
                    for (int i = 0; i < hiddenLayer; i++) {
                        cout << outputL1[i] << " ";
                    }
                    cout << endl;
                    cout << "biasHidden: " ;
                    for (int i = 0; i < hiddenLayer; i++) {
                        cout << biasHidden[i] << " ";
                    }
                    cout << endl;
                    cout << "biasOutput: " ;
                    for (int i = 0; i < outputLayer; i++) {
                        cout << biasOutput[i] << " ";
                    }
                    cout << endl;
                    cout << "weightHidden: " ;
                    // for (int i = 0; i < outputLayer; i++) {
                        cout << weightHidden[0] << " " << weightHidden[3]<< " " << weightHidden[4]<< " " << weightHidden[5]<< " " << weightHidden[6] << endl;
                    // }
                    cout << endl;  
                    cout << "weightOutput: " ;
                    // for (int i = 0; i < outputLayer; i++) {
                        cout << weightOutput[0] << " " << weightOutput[3]<< " " << weightOutput[4]<< " " << weightOutput[5]<< " " << weightOutput[6] << endl;
                    // }
                    cout << endl;  
                }
                
                
            }

            delete[] outputL1;
            delete[] outputL2;
        }
};


// both yTrue and yPred have size elements
double accuracy(int* yTrue, int* yPred, int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        cout << "yTrue " << yTrue[i] << " | " << "yPred " << yPred[i] << endl;
        if (yTrue[i] == yPred[i]) {
            sum++;
        }
    }
    return (sum / (1.0 * size));
}

int main() {
    cout << "start" << endl;
    int numSamples = 890;
    // double testSize = 0.3;
    double testSize = 0.25;
    double trainSize = 0.75;
    int numTrain = trainSize * numSamples;
    int numTest = testSize * numSamples;
    int numFeatures = NUM_FEATURES;

    int numHidden = 10; // 2/3 * 8 ~ 5.33 + 2 = 7.33
    int numClasses = NUM_CLASSES;
    int biasHiddenValue = -1;
    int biasOutputValue = -1;

    // SVM hyperparameters
    double iters = 1000;
    // double learningRate = 1.0 / iters; //1e-3
    double learningRate = 0.00005; //1e-3
    
    cout << "defined params" << endl;

    //allocate memory for training and test data
    double** Xtrain = new double*[numTrain];
    int* ytrain = new int[numTrain];
    for (int i = 0; i < numTrain; i++) {
        Xtrain[i] = new double[numFeatures];
    }
    
    double** Xtest = new double*[numTest];
    int* ytest = new int[numTest];
    for (int i = 0; i < numTest; i++) {
        Xtest[i] = new double[numFeatures];
    }

    double* Xtrain1D = new double[numTrain * numFeatures];
    double* Xtest1D = new double[numTest * numFeatures];
    
    cout << "finished allocation" << endl;

    // read from csv: https://www.youtube.com/watch?v=NFvxA-57LLA
    ifstream inputFile;
    // inputFile.open("titanic_prep.csv");
    // inputFile.open("titanic_prep_2.csv");
    inputFile.open("NEW_titanic_norm_cont_formatted.csv");
    if (!inputFile) {
        cout << "BAD FILE" << endl;
    }
    


    cout << "open file" << endl;

    string line = "";
    int total = 0;
    bool flag = true;
    int idx = 0;
    while (getline(inputFile, line)) {
        // cout << "inside while" << endl;
        if (flag) {
            flag = false;
            continue;
        }
        double xData1;
        double xData2;
        double xData3;
        double xData4;
        double xData5;
        double xData6;
        double xData7;
        double xData8;
        double xData9;
        int cls;
        int rowNum;
        string temp = "";

        stringstream inputString(line);
        getline(inputString, temp, ',');
        rowNum = atoi(temp.c_str());

        getline(inputString, temp, ',');
        xData1 = atof(temp.c_str());

        getline(inputString, temp, ',');
        cls = atoi(temp.c_str());

        getline(inputString, temp, ',');
        xData2 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData3 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData4 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData5 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData6 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData7 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData8 = atof(temp.c_str());

        getline(inputString, temp, ',');
        xData9 = atof(temp.c_str());

        if (total == numTrain) {
            idx = 0;
        }
        // cout << "cls = " << cls << endl;
        // cout << "total = " << total << " | numTrain = " << numTrain << " | numTest = " << numTest << " | idx = " << idx << endl;
        // cout << xData1 << ", " << xData2 << ", " << xData3 << ", " << xData4 << ", " << xData5 << ", " << xData6 << ", " << xData7 << ", " << xData8 << ", " << xData9 << " | cls = " << cls << endl;
        if (total < numTrain) {
            Xtrain[idx][0] = xData2;
            Xtrain[idx][1] = xData3;
            Xtrain[idx][2] = xData4;
            Xtrain[idx][3] = xData5;
            Xtrain[idx][4] = xData6;
            Xtrain[idx][5] = xData7;
            Xtrain[idx][6] = xData8;
            Xtrain[idx][7] = xData9;
            ytrain[idx] = cls;
        } else {
            // Xtest[idx][0] = xData1;
            cout << "iteration finished" << endl;
            Xtest[idx][0] = xData2;
            Xtest[idx][1] = xData3;
            Xtest[idx][2] = xData4;
            Xtest[idx][3] = xData5;
            Xtest[idx][4] = xData6;
            Xtest[idx][5] = xData7;
            Xtest[idx][6] = xData8;
            Xtest[idx][7] = xData9;
            ytest[idx] = cls;
        }

        line = "";
        total++;
        idx++;

        if (total == (numTrain + numTest)) {
            break;
        }
    }


    for (int i = 0; i < numTrain; i++) {
        for (int j = 0; j < numFeatures; j++) {
            Xtrain1D[(i * numFeatures) + j] = Xtrain[i][j];
        }
    }
    
    for (int i = 0; i < numTest; i++) {
        for (int j = 0; j < numFeatures; j++) {
            Xtest1D[(i * numFeatures) + j] = Xtest[i][j];
        }
    }
    
    cout << "file read" << endl;
    inputFile.close();
    cout << "file closed" << endl;

    MLP classifier = MLP(numFeatures, numHidden, numClasses, learningRate, iters, 
                            biasHiddenValue, biasOutputValue, numClasses);
    
    classifier.fit(Xtrain1D, ytrain, numTrain);
    cout << "classifier trained" << endl;

    int* predictions = new int[numTest * numClasses];
    classifier.predict(Xtest1D, predictions, numTest);
    cout << "predictions completed" << endl;

    int* predictedLabels = new int[numTest];
    for (int i = 0; i < numTest; i++) {
        if ((predictions[(i * numClasses) + 0] == 1) && (predictions[(i * numClasses) + 1] == 0)) {
            predictedLabels[i] = 0;
        } else if ((predictions[(i * numClasses) + 0] == 0) && (predictions[(i * numClasses) + 1] == 1)) {
            predictedLabels[i] = 1;
        } else {
            predictedLabels[i] = -1;
            cout << "YIKES! p[0] = " << predictions[(i * numClasses) + 0] << ", p[1] = " << predictions[(i * numClasses) + 1] << endl;
        }
    }

    double acc = accuracy(ytest, predictedLabels, numTest);
    printf("MLP Accuracy: %f\n", acc);

    delete[] predictions;
    delete[] predictedLabels;
    delete[] Xtrain1D;
    delete[] Xtest1D;
    
    //free memory of training and test data
    for (int i = 0; i < numTrain; i++) {
        delete[] Xtrain[i];
    }
    delete[] Xtrain;
    delete[] ytrain;
    
    for (int i = 0; i < numTest; i++) {
        delete[] Xtest[i];
    }
    delete[] Xtest;
    delete[] ytest;
}