// https://towardsdatascience.com/implementing-svm-from-scratch-784e4ad0bc6a
// Implementation based off of python SVM techniques from this medium article

// Basic setup/initialization of weights and biases (w, b)
// Map the class labels from {0,1} to {-1,1}
// Perform gradient descent for n iterations, which involves the computation
//      of gradients and updated the weights an biases accordingly
// Make the final prediction


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>

using namespace std;

#define NUM_FEATURES 2
#define NUM_PIXELS 2

//NUM_FEATURES = lenW = 2
// kernel<<<dimGrid, (numIters, size, lenW, gpuW, b, gpuCLS, gpuX, lr, lambdaParam);				
/*
__global__ void kernel(int numIters, int size, int lenW, double* w, double* b, double* cls, double* X, double lr, double lambdaParam) {

    int id = threadIdx.x;
    printf("id = %d\n", id);
    __shared__ double myW[NUM_FEATURES];
    __shared__ bool constraint;
    __shared__ double myB;
    __shared__ double x[NUM_FEATURES];
    myW[id] = w[id];
    myB = b[0];
    // if (id == 0) {
    //     printf("id = 0, myB = %f\n",myB);
    // }
    double result = 0;
    double xVal = 0;
    for (int i = 0; i < numIters; i++) {
        for (int j = 0; j < size; j++) {
            //check constraint: dotProduct with recursive doubling
            xVal = X[(j * NUM_FEATURES) + id];
            x[id] = xVal * myW[id];
            // if ((id == 0) && (i % 100 == 0) && (j == 0)) {
            //     printf("id = 0, i = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", i, myW[id], xVal, cls[j]);
            // }
            for (int k = 2; k <= lenW; k *= 2) {
                __syncthreads();
                if ((id % k) == 0) {
                    x[id] += x[id + (k/2)];
                }
            }

            if (id == 0) {
                result = x[id] + myB;
                // if ((i % 100 == 0) && (j == 0)) {
                //     printf("id = 0, i = %d, result = %f\n", i, result);
                // }
                if ((cls[j] * result) >= 1) {
                    constraint = true;
                    // myB = 0;
                } else {
                    constraint = false;
                    myB -= lr * (-cls[j]);
                }
                // if ((i % 100 == 0) && (j == 0)) {
                //     printf("id = 0, i = %d, constraint = %d\n", i, constraint);
                // }
                
            }

            //get and update gradients
            if (constraint) {
                myW[id] = myW[id] - (lr * myW[id] * lambdaParam);
                // if ((id == 0) && (i % 100 == 0) && (j == 0)) {
                //     printf("constrain: myW[id] = %f\n", myW[id]);
                // }
            } else {
                myW[id] = myW[id] - (lr * ((lambdaParam * myW[id]) - (cls[j] * xVal)));
                // if ((id == 0) && (i % 100 == 0) && (j == 0)) {
                //     printf("else constrain: myW[id] = %f\n", myW[id]);
                // }
            }
            
        }
        __syncthreads();
    }
    
    w[id] = myW[id];
    if (id == 0) {
        b[0] = myB;
        // printf("end w[id] = %f, b = %f\n", w[id], b[0]);
    }
}
*/

__global__ void kernel(int numIters, int size, int lenW, double* w, double* b, double* cls, double* X, double lr, double lambdaParam) {

    int id = threadIdx.x;
    __shared__ double myW[NUM_PIXELS];
    __shared__ bool constraint;
    __shared__ double myB;
    __shared__ double x[NUM_PIXELS];

    myW[id] = 0;
    myB = 0;
    // if (id == 0) {
    // printf("id = %d, digit = %d, myB = %f\n", id, digit, myB);
    // }

    // for (int i = 0; i < 8400; i++) {
    //     if (i % 1000 == 0) {
    //         printf("id = %d, i = %d, X[i] = %f\n", id, i, X[i]);
    //     }
    // }

    double result = 0;
    double xVal = 0;
    for (int i = 0; i < numIters; i++) {
        for (int j = 0; j < size; j++) {
            //check constraint: dotProduct with recursive doubling
            
            xVal = X[(j * NUM_PIXELS) + id];

            // xVal = X[(j * BAD_PIXELS) + id];
            x[id] = xVal * myW[id];
            __syncthreads();
            /*
            for (int k = 2; k <= MORE_PIXELS; k *= 2) {
            // for (int k = 2; k <= BAD_PIXELS; k *= 2) {
                __syncthreads();
                if ((id % k) == 0) {
                    // printf("DOUBLING k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);

                    if ((id + (k/2)) < NUM_PIXELS) {
                        x[id] += x[id + (k/2)];
                        // printf("DOUBLING k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);
                    } 
                    // else {
                    //     printf("NOOOOO k = %d, digit = %d, id = %d, i = %d, j = %d, myW[id] = %f, xVal = %f, cls[j] = %f\n", k, digit, id, i, j, myW[id], xVal, cls[j]);
                    // }
                }
            }
            */

            if (id == 0) {
                // result = x[id] + myB;
                result = 0;
                for (int k = 0; k < NUM_PIXELS; k++) {
                    result += x[k];
                }
                result += myB;

                if ((cls[j] * result) >= 1) {
                    constraint = true;
                } else {
                    constraint = false;
                    myB -= lr * (-cls[j]);
                }
                
            }
            __syncthreads();

            //get and update gradients
            if (constraint) {
                myW[id] = myW[id] - (lr * myW[id] * lambdaParam);
            } else {
                myW[id] = myW[id] - (lr * ((lambdaParam * myW[id]) - (cls[j] * xVal)));
            }
            
        }
        __syncthreads();
    }
    
    w[id] = myW[id];
    if (id == 0) {
        b[0] = myB;
    }
    // printf("digit = %d, id = %d, end w[id] = %f, b = %f\n", digit, id, w[id], b[0]);
    
}


class SVM {

    private:
        double lr;
        double lambdaParam;
        int numIters;
        double* w;
        int lenW;
        double b;
        double* clsMap;

            
        void initWeightsBias(int numFeatures) { //self, X
            w = new double[numFeatures];
            for (int i = 0; i < numFeatures; i++) {
                w[i] = 0;
            }
            lenW = numFeatures;
            b = 0;
        }

        void getClsMap(int* y, int size) { //self, y
            if (clsMap == NULL) {
                clsMap = new double[size];
            }
            for (int i = 0; i < size; i++) {
                if (y[i] == 0) {
                    clsMap[i] = -1;
                } else {
                    clsMap[i] = 1;
                }
            }
        }

        // a and b must be the same length
        double dotProduct(double* a, double* b, int size) {
            double sum = 0;
            for (int i = 0; i < size; i++) {
                if (i % 100 == 0) {
                    cout << "dot product i = " << i << ", a[i] = " << a[i] << ", b[i] = " << b[i] << endl;
                }
                sum += a[i] * b[i];
            }
            return sum;
        }

        // GPU
        bool satisfyConstraint(double* x, int idx) { //self, x, idx
            // dot product linearModel = x * w + b
            double linearModel = dotProduct(x, w, lenW);
            linearModel += b;

            if ( (clsMap[idx] * linearModel) >= 1) {
                return true;
            }
            return false;
        }

        // FREE dw
        void getGradients(bool constraint, double* x, int idx, double*& dw, double& db) { //self, constrain, x, idx
            dw = new double[lenW];
            if (constraint) {
                for (int i = 0; i < lenW; i++) {
                    dw[i] = (w[i] * lambdaParam);
                }
                db = 0;
                return;
            }

            for (int i = 0; i < lenW; i++) {
                dw[i] = (lambdaParam * w[i]) - (clsMap[idx] * x[i]);
            }
            db = -clsMap[idx];
        }

        void updateWeights(double* dw, double db) { //self, dw, db
            for (int i = 0; i < lenW; i++) {
                w[i] -= lr * dw[i];
            }
            b -= lr * db;
        } 

    public:
        SVM(double learningRate, double lamba, double iters) {
            lr = learningRate;
            lambdaParam = lamba;
            numIters = iters;
            w = NULL;
            b = 0;
            clsMap = NULL;
        }

        ~SVM() {
            if (w != NULL) {
                printf("free memory w\n");
                delete[] w;
            }
            if (clsMap != NULL) {
                printf("free memory clsMap\n");
                delete[] clsMap;
            }
        }

        //size = length of y, length of X (X dims = [size][numFeatures])
        void fit(double* X, int numFeatures, int* y, int size) { //self, X, y
            
            initWeightsBias(numFeatures);
            getClsMap(y, size); // updates clsMap

            double* gpuX;
            double* gpuCLS;
            double* gpuW;
            double* gpuB;

            double* bArr = new double[1];
            bArr[0] = b;

	        hipMalloc((void**)&gpuX, sizeof(double)*size*numFeatures); 
	        hipMalloc((void**)&gpuCLS, sizeof(double)*size); 
	        hipMalloc((void**)&gpuW, sizeof(double)*lenW);  
	        hipMalloc((void**)&gpuB, sizeof(double)); 
            struct timespec start, stop; 
            double time;
            if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
            
	        hipMemcpy(gpuX, X, sizeof(double)*size*numFeatures, hipMemcpyHostToDevice);
	        hipMemcpy(gpuCLS, clsMap, sizeof(double)*size, hipMemcpyHostToDevice);
	        hipMemcpy(gpuW, w, sizeof(double)*lenW, hipMemcpyHostToDevice);
            hipMemcpy(gpuB, bArr, sizeof(double), hipMemcpyHostToDevice);

            //kernel(int numIters, int size, int lenW, double* w, double b, double* cls, double* X) {
            // cout << "lenW = " << lenW << endl;
            dim3 dimGrid(1);
            dim3 dimBlock(lenW);
            	
            kernel<<<dimGrid, dimBlock>>>(numIters, size, lenW, gpuW, gpuB, gpuCLS, gpuX, lr, lambdaParam);				
 
            
            // cudaMemcpy(X, gpuX, sizeof(double)*size*numFeatures, cudaMemcpyDeviceToHost);
	        // cudaMemcpy(clsMap, gpuCLS, sizeof(double)*size, cudaMemcpyDeviceToHost);
	        hipMemcpy(w, gpuW, sizeof(double)*lenW, hipMemcpyDeviceToHost);
	        hipMemcpy(bArr, gpuB, sizeof(double), hipMemcpyDeviceToHost);

            if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	  
            time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
            printf("time is %f ns\n", time*1e9);	
            
            b = bArr[0];
            // printf("after kernel bArr = %f, b = %f\n", bArr[0], b);
            delete[] bArr;

            hipFree(gpuX);  
            hipFree(gpuCLS);  
            hipFree(gpuW);  
            hipFree(gpuB);

        }

        //X is of dims [size][numFeatures]
        // FREE estimate
        int* predict(double** X, int size) { //self, X
            // printf("predict w[id] = %f, b = %f\n", w[0], b);
            int* estimate = new int[size]; //hold dot products
            for (int i = 0; i < size; i++) {
                estimate[i] = dotProduct(X[i], w, lenW);
                estimate[i] += b;
            }

            //predict class labels
            for (int i = 0; i < size; i++) {
                if (estimate[i] < 0) {
                    estimate[i] = 0;
                } else {
                    estimate[i] = 1;
                }
            }
            return estimate;
        }

};

// both yTrue and yPred have size elements
double accuracy(int* yTrue, int* yPred, int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        cout << "yTrue " << yTrue[i] << " | " << "yPred " << yPred[i] << endl;
        if (yTrue[i] == yPred[i]) {
            sum++;
        }
    }
    return (sum / (1.0 * size));
}

void parseBlobData(string dataFileStr, double** Xtrain, int* ytrain, int numTrain, double** Xtest, int* ytest) {
    ifstream inputFile;
    inputFile.open(dataFileStr);
    cout << "open file" << endl;
    
    string line = "";
    int total = 0;
    bool flag = true;
    int idx = 0;
    while (getline(inputFile, line)) {
        if (flag) {
            flag = false;
            continue;
        }
        double xData1;
        double xData2;
        int cls;
        string temp = "";

        stringstream inputString(line);
        // ss >> xData1 >> xData2 >> cls;
        getline(inputString, temp, ',');
        xData1 = atof(temp.c_str());
        getline(inputString, temp, ',');
        xData2 = atof(temp.c_str());
        getline(inputString, temp, ',');
        cls = atoi(temp.c_str());

        if (total == numTrain) {
            idx = 0;
        }
        cout << "total = " << total << " | numTrain = " << numTrain << " | idx = " << idx << endl;
        cout << "xData1 = " << xData1 << " | xData2 = " << xData2 << " | cls = " << cls << endl;
        if (total < numTrain) {
            Xtrain[idx][0] = xData1;
            Xtrain[idx][1] = xData2;
            ytrain[idx] = cls;
        } else {
            Xtest[idx][0] = xData1;
            Xtest[idx][1] = xData2;
            ytest[idx] = cls;
        }

        line = "";
        total++;
        idx++;

    }
        
    cout << "file read" << endl;
    inputFile.close();
    cout << "file closed" << endl;

}

// FINISH THIS?
/*
void parseTitanicData(string dataFileStr, double** Xtrain, double* ytrain, , double** Xtest, double* ytest) {
    ifstream inputFile;
    inputFile.open(dataFileStr);
    cout << "open file" << endl;
    
    string line = "";
    int total = 0;
    bool flag = true;
    int idx = 0;
    while (getline(inputFile, line)) {
        if (flag) {
            flag = false;
            continue;
        }
        int p1;
        int survived;
        int pclass;
        // string name;
        string sex;
        int age;
        int sibSp;
        int parch;
        string ticket;
        double fare;
        string cabin;
        string embarked;

        string temp = "";

        stringstream inputString(line);
        // ss >> xData1 >> xData2 >> cls;
        getline(inputString, temp, ',');
        p1 = atoi(temp.c_str());
        getline(inputString, temp, ',');
        survived = atof(temp.c_str());
        getline(inputString, temp, ',');
        pclass = atoi(temp.c_str());

        

        if (total == numTrain) {
            idx = 0;
        }
        cout << "total = " << total << " | numTrain = " << numTrain << " | numTest = " << numTest << " | idx = " << idx << endl;
        cout << "xData1 = " << xData1 << " | xData2 = " << xData2 << " | cls = " << cls << endl;
        if (total < numTrain) {
            Xtrain[idx][0] = xData1;
            Xtrain[idx][1] = xData2;
            ytrain[idx] = cls;
        } else {
            Xtest[idx][0] = xData1;
            Xtest[idx][1] = xData2;
            ytest[idx] = cls;
        }

        line = "";
        total++;
        idx++;

    }
        
    cout << "file read" << endl;
    inputFile.close();
    cout << "file closed" << endl;

}
*/

int main() {

    cout << "start" << endl;
    // training/test data parameters
    int numSamples = 250;
    double testSize = 0.1;
    int numTrain = (1 - testSize) * numSamples;
    int numTest = testSize * numSamples;
    int numFeatures = 2;

    // SVM hyperparameters
    double learningRate = 0.001; //1e-3
    double lamba = 0.01; //1e-2 
    double iters = 1000;
    
    cout << "defined params" << endl;

    //allocate memory for training and test data
    double** Xtrain = new double*[numTrain];
    int* ytrain = new int[numTrain];
    for (int i = 0; i < numTrain; i++) {
        Xtrain[i] = new double[numFeatures];
    }
    
    double** Xtest = new double*[numTest];
    int* ytest = new int[numTest];
    for (int i = 0; i < numTest; i++) {
        Xtest[i] = new double[numFeatures];
    }

    
    cout << "finished allocation" << endl;
    cout << "numTrain = " << numTrain << " | numTest = " << numTest << endl;

    // read from csv: https://www.youtube.com/watch?v=NFvxA-57LLA
    string dataFileStr = "blob_data.csv";

    if (dataFileStr == "blob_data.csv") {
        parseBlobData(dataFileStr, Xtrain, ytrain, numTrain, Xtest, ytest);
    } else if (dataFileStr == "titanic.csv") {
        // parseTitanicData(dataFileStr, Xtrain, ytrain, Xtest, ytest);
    } else {
        cout << "File " << dataFileStr << " not supported" << endl;
    }

    double* Xtrain1D = new double[numTrain * numFeatures];
    for (int i = 0; i < numTrain; i++) {
        for (int j = 0; j < numFeatures; j++) {
            Xtrain1D[(i * numFeatures) + j] = Xtrain[i][j];
        }
    }

    SVM classifier = SVM(learningRate, lamba, iters);
    classifier.fit(Xtrain1D, numFeatures, ytrain, numTrain);
    int* predictions = classifier.predict(Xtest, numTest);
    
    
    cout << "cassifier trained " << endl;
    double acc = accuracy(ytest, predictions, numTest);
    printf("SVM Accuracy: %f\n", acc);

    delete[] predictions;
    delete[] Xtrain1D;

    //free memory of training and test data
    for (int i = 0; i < numTrain; i++) {
        delete[] Xtrain[i];
    }
    delete[] Xtrain;
    delete[] ytrain;
    
    for (int i = 0; i < numTest; i++) {
        delete[] Xtest[i];
    }
    delete[] Xtest;
    delete[] ytest;

    cout << "done" << endl;
    
}